#include <iostream>
#include <filesystem>
#include <hip/hip_runtime.h>

__global__ void noop() {}

int main() {
    // Demonstrate CUDA context (will report if not available)
    noop<<<1,1>>>();
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA not available or error: " << hipGetErrorString(err) << "\n";
    }

    try {
        for (const auto& entry : std::filesystem::directory_iterator(".")) {
            std::cout << entry.path().filename().string() << '\n';
        }
    } catch (const std::exception& e) {
        std::cerr << "Error listing directory: " << e.what() << '\n';
        return 1;
    }
    return 0;
}
