#include <iostream>
#include <filesystem>
#include <hip/hip_runtime.h>

__global__ void noop() {}

int main(int argc, char** argv) {
    // Demonstrate CUDA context (will report if not available)
    noop<<<1,1>>>();
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA not available or error: " << hipGetErrorString(err) << "\n";
    }

    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <source> <destination>\n";
        return 2;
    }

    try {
        std::filesystem::copy_file(
            argv[1],
            argv[2],
            std::filesystem::copy_options::overwrite_existing
        );
    } catch (const std::exception& e) {
        std::cerr << "Copy failed: " << e.what() << '\n';
        return 1;
    }
    return 0;
}
